#include "hip/hip_runtime.h"
/* ---------------------------------------------------
   My Hello world for CUDA programming
   --------------------------------------------------- */

#include <stdio.h>        // C programming header file
#include <unistd.h>       // C programming header file
#include <hip/hip_runtime.h>
#include "hello_cuda.cuh"                            // cude.h is automatically included by nvcc...

inline void __checkMsg(hipError_t code, const char *file, const int line)
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err)
  {
    fprintf(stderr, "checkMsg() CUDA error: %s in file <%s>, line %i : %s.\n", hipGetErrorString(code), file, line, hipGetErrorString(err));
    exit(-1);
  }
}
inline void __checkMsgNoFail(hipError_t code, const char *file, const int line)
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err)
  {
    fprintf(stderr, "checkMsg() CUDA warning: %s in file <%s>, line %i : %s.\n", hipGetErrorString(code), file, line, hipGetErrorString(err));
  }
}

/* ------------------------------------
   Your first kernel (= GPU function)
   ------------------------------------ */
__global__ void hello( )
{
   printf("Hello World GPU!\n");
}

void start_hello()
{

   hello<<< 1, 4 >>>( );

   hipDeviceProp_t prop;
   checkMsg(hipGetDeviceProperties(&prop,0));

   printf("I am the CPU: Hello World ! \n");

   sleep(1);   // Necessary to give time to let GPU threads run !!!

}
